#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <hip/hip_runtime.h>
#include "operator.h"
#include "strcut.h"
#include "hipblas.h"

__device__ double getPearson(double* d_dataList, int row, int col, int conNum)
{
    int num = 0;
    double mulE = 0, yDataE = 0, xDataE = 0;
    for (int i = 0; i < conNum; i++) {
        if (d_dataList[row * conNum + i] == 0 || d_dataList[col * conNum + i] == 0) {
            num++;
            continue;
        }
        mulE += d_dataList[row * conNum + i] * d_dataList[col * conNum + i];
        xDataE += d_dataList[row * conNum + i];
        yDataE += d_dataList[col * conNum + i];
    }
    double numerator = mulE - (xDataE * yDataE / (conNum - num));
    num = 0;
    double	xDataSquareAdd = 0, xDataAdd = 0, yDataSquareAdd = 0, yDataAdd = 0;
    for (int i = 0; i < conNum; i++) {
        if (d_dataList[row * conNum + i] == 0 || d_dataList[col * conNum + i] == 0) {
            num++;
            continue;
        }
        xDataSquareAdd += d_dataList[row * conNum + i] * d_dataList[row * conNum + i];
        xDataAdd += d_dataList[row * conNum + i];
        yDataSquareAdd += d_dataList[col * conNum + i] * d_dataList[col * conNum + i];
        yDataAdd += d_dataList[col * conNum + i];
    }
    double denominator = sqrt((xDataSquareAdd - xDataAdd * xDataAdd / (conNum - num)) * (yDataSquareAdd - yDataAdd * yDataAdd / (conNum - num)));
    return  fabs(numerator / denominator);
}

__global__ void kernelPearson(double* d_dataList, double* d_coeMatrix, int num, int conNum)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num * num)
        return;
    int row = index / num;
    int col = index % num;
    if (row > col)
        return;
    double pearson = getPearson(d_dataList, row, col, conNum);
    d_coeMatrix[row * num + col] = pearson;
    d_coeMatrix[col * num + row] = pearson;

}

__global__ void kernelTransMatrix(double* d_coeMatrix, double* d_coeSum, double* d_transMatrix, int num, int highNum, int* d_numList) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num * num)
        return;
    int row = index / num;
    int col = index % num;
    if (col == num - 1) {
        d_transMatrix[index] = 1.0 / d_coeSum[row];
    }
    else if (row == num - 1)
    {
        d_transMatrix[index] = double(d_numList[col]) / highNum;
    }
    else {
        d_transMatrix[index] = d_coeMatrix[row * (num - 1) + col] / d_coeSum[row];
    }
    if (row == col)
        d_transMatrix[index] = 0.0;
}

__global__ void kernelLrValue(double* d_lrValue, double* d_transMatrix, double* d_nextLrValue, int num, double* d_error) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num)
        return;
    d_nextLrValue[index] = 0;
    for (int i = 0; i < num; i++)
    {
        d_nextLrValue[index] += d_lrValue[i] * d_transMatrix[i * num + index];
    }
    d_error[index] = abs(d_nextLrValue[index] - d_lrValue[index]);
}

__global__ void kernelLrValueEq(double* d_lrValue, double* d_nextLrValue, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num)
        return;
    d_lrValue[index] = d_nextLrValue[index];
}

__global__ void kernelLrValueEq2(double* d_lrValue, double* d_nextLrValue, int num, double* d_error) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num)
        return;
    d_error[index] = abs(d_nextLrValue[index] - d_lrValue[index]);
    d_lrValue[index] = d_nextLrValue[index];
}

__global__ void kernelLrValueShare(double* d_lrValue, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num - 1)
        return;
    d_lrValue[index] += d_lrValue[num - 1] / (num - 1);
}

__global__ void kernelFunction1(double* d_coeMatrix, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num * num)
        return;
    if (index % num != index / num)
        d_coeMatrix[index] = d_coeMatrix[index] * d_coeMatrix[index] * d_coeMatrix[index];
}

__global__ void kernelFunction2(double* d_coeMatrix, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num * num)
        return;
    if (index % num != index / num)
        d_coeMatrix[index] = d_coeMatrix[index] * (d_coeMatrix[index] + 0.5);
}

__global__ void kernelFunction3(double* d_coeMatrix, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num * num)
        return;
    if (index % num != index / num)
        d_coeMatrix[index] = d_coeMatrix[index] * (d_coeMatrix[index] + 1.0);
}

__global__ void kernelPearsonCut(double* d_coeMatrix, int num, double cutThreshold) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num * num)
        return;
    if (d_coeMatrix[index] > cutThreshold)
    {
        d_coeMatrix[index] = 1;
    }
    else
    {
        d_coeMatrix[index] = 0;
    }
}

__global__ void kernelGround(double* d_corMatrix, int num, int* d_numList)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num)
        return;
    int highCoeNum = 1;
    for (int i = 0; i < num; i++)
    {
        if (d_corMatrix[index * num + i] > 0.8)
            highCoeNum++;
    }
    d_numList[index] = highCoeNum;
}

__global__ void kernelTransMatrixD(double* d_coeMatrix, double* d_coeSum, double* d_transMatrix, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > num * num)
        return;
    int row = index / num;
    int col = index % num;
    if (col == num - 1) {
        d_transMatrix[index] = 1.0 / d_coeSum[row];
    }
    else if (row == num - 1)
    {
        d_transMatrix[index] = 1.0 / (num - 1);
    }
    else {
        d_transMatrix[index] = d_coeMatrix[row * (num - 1) + col] / d_coeSum[row];
    }
    if (row == col)
        d_transMatrix[index] = 0.0;
}


Operator::Operator(const std::string outPath, const std::string inputPath) : outPath(outPath), inputPath(inputPath)
{
}

Operator::~Operator() {
    hipFree(d_transMatrix);
    hipFree(d_coeMatrix);
    delete[]lrValue;
}

void Operator::quickSort(int low, int high, int array[], double* res)
{
    int Low, High, temp;
    if (low < high) {
        Low = low;
        High = high;
        temp = array[low];
        while (Low < High) {
            while (Low < High && res[array[High]] >= res[temp]) {
                High--;
            }
            if (Low < High) {
                array[Low] = array[High];
                Low++;
            }
            while (Low < High && res[array[Low]] <= res[temp]) {
                Low++;
            }
            if (Low < High) {
                array[High] = array[Low];
                High--;
            }
        }
        array[Low] = temp;
        quickSort(low, Low - 1, array, res);
        quickSort(Low + 1, high, array, res);
    }
}


void Operator::printMyGenes() {
    int num = myGenes.size();
    for (unsigned int i = 0; i < myGenes.size(); i++)
    {
        std::cout << myGenes[i].name << ",";
    }
    std::cout << std::endl;
    /*for (int i = 0; i < num*conNum; i++)
    {
        if (i % conNum == 0)
            std::cout << std::endl;
        std::cout << dataList[i] << " ";
    }*/

}

void Operator::printCoeMatrix() {
    int num = myGenes.size();
    std::string outPathCoe = outPath + "_coe.txt";
    std::ofstream outFile(outPathCoe, std::ios::ate);
    for(int i = 0; i < num; i++)
    {
     outFile << "," << myGenes[i].name;
    }
    int id=0; 
    for (int i = 0; i < num * num; i++)
    {
        if (i % num == 0){
          outFile << "\n";
          outFile << myGenes[id++].name;
        }
        //std::cout<< coeMatrix[i];
        outFile  << "," << coeMatrix[i];
    }
}

void Operator::cutAndOut() {
    int num = myGenes.size();
    std::string outPathCoe = outPath + "_cut.txt";
    std::ofstream outFile(outPathCoe, std::ios::ate);

    for (int i = 0; i < num * num; i++)
    {   
        int j = i / num;
        int k = i % num;
        if (k<=j)
            continue;
        if (coeMatrix[i]){
          outFile << myGenes[j].name << "," << myGenes[k].name << "\n";
          }
    }
}


void Operator::outputFile(const int coeFlag,double cutThreshold) {
    int num = myGenes.size() - 1;
    std::string outPathLr = outPath + "_lr.txt";
    std::ofstream outFile(outPathLr, std::ios::ate);
    if (!outFile)
    {
        std::cout << "Error!" << std::endl;
        exit(1);
    }
    int* mediate = new int[num];
    for (int i = 0; i <= num; i++) {
        mediate[i] = i;
    }
    double* res = lrValue;
    quickSort(0, num, mediate, lrValue);
    int flag = 0;
    for (int i = num; i >= 0; i--)
    {
        outFile << myGenes[mediate[i]].name << " = " << res[mediate[i]] << std::endl;
        if (flag < 15)
        {
            idList.push_back(mediate[i]);
            flag++;
        }
    }
    outFile.close();
    if(coeFlag){
        num = myGenes.size();
        coeMatrix = new double[num * num]();
        hipMemcpy(coeMatrix, d_coeMatrix, num * num * sizeof(double), hipMemcpyDeviceToHost);
        printCoeMatrix();
    }
    if(cutThreshold){
        dim3 blockSize(256);
        dim3 gridSize((num * num + blockSize.x - 1) / blockSize.x);
        kernelPearsonCut << <gridSize, blockSize >> > (d_coeMatrix, num, cutT);
        num = myGenes.size();
        coeMatrix = new double[num * num]();
        hipMemcpy(coeMatrix, d_coeMatrix, num * num * sizeof(double), hipMemcpyDeviceToHost);
        cutAndOut();
    }

}

void Operator::printTransMatrix() {
    int num = myGenes.size() + 1;
    for (int i = 0; i < num; i++)
    {
        double sum = 0;
        for (int j = 0; j < num; j++)
        {
            sum += transMatrix[i * num + j];
        }
        std::cout << i << ":" << sum << std::endl;
    }
    for (int i = 0; i < num; i++)
    {
        for (int j = 0; j < num; j++)
        {
            std::cout << transMatrix[i * num + j] << " ";
        }
        std::cout << std::endl;
    }
}

void Operator::readFileCoe() {
    int num;
    std::ifstream fp(inputPath);
    std::string strLine;
    int a = -1;
    while (std::getline(fp, strLine))
    {
        a++;
    }
    conNum = a;
    fp.clear();
    fp.seekg(std::ios::beg);

    int index = -1;
    while (std::getline(fp, strLine)) {
        std::vector<std::string> values;
        splitStr(strLine, ",", values);
        if (index == -1)
        {
            for (unsigned int k = 0; k < values.size(); ++k)
            {
                geneNode item;
                item.id = k;
                item.name = values[k];
                myGenes.push_back(item);
            }
            num = myGenes.size();
            dataList = new double[num * conNum]();
        }
        else
        {
            for (unsigned int k = 0; k < values.size(); ++k)
            {
                auto fvalue = atof(values[k].c_str());
                dataList[k * conNum + index] = fvalue;
            }
        }
        index++;
    }
    lrValue = new double[num + 1];
    for (int i = 0; i < num + 1; i++)
    {
        lrValue[i] = 1.0;
    }
    lrValue[num] = 0.0;
    fp.close();
    int len = myGenes[num-1].name.length();
    myGenes[num-1].name=myGenes[num-1].name.substr(0,len-1);
    hipMalloc(&d_dataList, num * conNum * sizeof(double));
    hipMemcpy(d_dataList, dataList, num * conNum * sizeof(double), hipMemcpyHostToDevice);
    delete[]dataList;
    std::cout<<num<<std::endl;
}

void Operator::readFileDegree() {
    std::ifstream fp(inputPath);
    std::string strLine;
    int index = -1;
    int num;
    while (std::getline(fp, strLine)) {
        std::vector<double> item;
        std::vector<std::string> values;
        splitStr(strLine, ",", values);
        if (index == -1)
        {
            for (unsigned int k = 0; k < values.size(); ++k)
            {
                geneNode item;
                item.id = k;
                item.name = values[k];
                if(k==values.size()-1)
                {
                  item.name = values[k].substr(0,values[k].length()-1);
                }
                myGenes.push_back(item);
            }
            num = myGenes.size();
            coeMatrix = new double[num * num]();
        }
        else
        {
            for (unsigned int k = 1; k < values.size(); ++k)
            {
                double fvalue = atof(values[k].c_str());
                coeMatrix[index * num + k - 1] = fvalue;
            }
        }
        index++;
    }
    lrValue = new double[num + 1]();
    for (int i = 0; i < num; i++)
    {
        lrValue[i] = 1.0;
    }
    fp.close();
    hipMalloc(&d_coeMatrix, num * num * sizeof(double));
    hipMemcpy(d_coeMatrix, coeMatrix, num * num * sizeof(double), hipMemcpyHostToDevice);
}

void Operator::calPearson() {

    int num = myGenes.size();
    hipMalloc(&d_coeMatrix, num * num * sizeof(double));
    dim3 blockSize(256);
    dim3 gridSize((num * num + blockSize.x - 1) / blockSize.x);
    kernelPearson << <gridSize, blockSize >> > (d_dataList, d_coeMatrix, num, conNum);
    hipFree(d_dataList);
}

void Operator::calTransMatrix()
{
    int num = myGenes.size();
    double* coeSum = new double[num + 1]();
    for (int i = 0; i < num; ++i)
    {
        double* d_rowSum;
        hipMalloc(&d_rowSum, num * sizeof(double));
        int nBegin = i * num;
        hipMemcpy(d_rowSum, d_coeMatrix + nBegin, sizeof(double) * num, hipMemcpyDeviceToDevice);
        thrust::device_ptr<double> dev_ptr(d_rowSum);
        double row_sum = thrust::reduce(dev_ptr, dev_ptr + num);
        hipFree(d_rowSum);
        coeSum[i] = row_sum;
    }
    coeSum[num] = 1;
    double* d_coeSum;
    hipMalloc(&d_coeSum, (num + 1) * sizeof(double));
    hipMemcpy(d_coeSum, coeSum, (num + 1) * sizeof(double), hipMemcpyHostToDevice);
    num++;
    transMatrix = new double[num * num];
    hipMalloc(&d_transMatrix, num * num * sizeof(double));
    dim3 blockSize(256);
    dim3 gridSize((num * num + blockSize.x - 1) / blockSize.x);
    kernelTransMatrix << <gridSize, blockSize >> > (d_coeMatrix, d_coeSum, d_transMatrix, num, highNum, d_numList);
    hipMemcpy(transMatrix, d_transMatrix, num * num * sizeof(double), hipMemcpyDeviceToHost);
    delete[]coeSum;
    hipFree(d_coeSum);
    hipFree(d_numList);
}

void Operator::calTransMatrixD()
{
    int num = myGenes.size();
    double* coeSum = new double[num]();
    for (int i = 0; i < num; ++i)
    {
        double* d_rowSum;
        hipMalloc(&d_rowSum, num * sizeof(double));
        int nBegin = i * num;
        hipMemcpy(d_rowSum, d_coeMatrix + nBegin, sizeof(double) * num, hipMemcpyDeviceToDevice);
        thrust::device_ptr<double> dev_ptr(d_rowSum);
        double row_sum = thrust::reduce(dev_ptr, dev_ptr + num);
        hipFree(d_rowSum);
        coeSum[i] = row_sum;
    }
    double* d_coeSum;
    hipMalloc(&d_coeSum, num * sizeof(double));
    hipMemcpy(d_coeSum, coeSum, num * sizeof(double), hipMemcpyHostToDevice);
    num++;
    transMatrix = new double[num * num];
    hipMalloc(&d_transMatrix, num * num * sizeof(double));
    dim3 blockSize(256);
    dim3 gridSize((num * num + blockSize.x - 1) / blockSize.x);
    kernelTransMatrixD << <gridSize, blockSize >> > (d_coeMatrix, d_coeSum, d_transMatrix, num);
    hipMemcpy(transMatrix, d_transMatrix, num * num * sizeof(double), hipMemcpyDeviceToHost);
    delete[]coeSum;
    hipFree(d_coeSum);
}

void Operator::calLrValue()
{
    int num = myGenes.size() + 1;
    int numIter = 1;
    double error;
    double* d_error;
    double* d_nextLrValue;
    double* d_lrValue;
    hipMalloc(&d_error, num * sizeof(double));
    hipMalloc(&d_lrValue, num * sizeof(double));
    hipMalloc(&d_nextLrValue, num * sizeof(double));
    hipMemcpy(d_lrValue, lrValue, num * sizeof(double), hipMemcpyHostToDevice);
    dim3 blockSize(256);
    dim3 gridSize((num + blockSize.x - 1) / blockSize.x);
    do
    {
        kernelLrValue << <gridSize, blockSize >> > (d_lrValue, d_transMatrix, d_nextLrValue, num, d_error);
        thrust::device_ptr<double> dev_ptr(d_error);
        error = thrust::reduce(dev_ptr, dev_ptr + num);
        kernelLrValueEq << <gridSize, blockSize >> > (d_lrValue, d_nextLrValue, num);
        //std::cout << numIter++ << std::endl;
        //std::cout << error << std::endl;
    } while (error > minError || numIter < maxNum);
    kernelLrValueShare << <gridSize, blockSize >> > (d_lrValue, num);
    hipMemcpy(lrValue, d_lrValue, num * sizeof(double), hipMemcpyDeviceToHost);
    /*
    num = num -1;
    coeMatrix = new double[num * num]();
    hipMemcpy(coeMatrix, d_coeMatrix, num * num * sizeof(double), hipMemcpyDeviceToHost);
    */
    hipFree(d_lrValue);
    hipFree(d_nextLrValue);
    hipFree(d_error);
}


void Operator::calIteration(int modId,double cutThreshold) {
    int num = myGenes.size();
    hipMalloc(&d_numList, num * sizeof(int));
    dim3 blockSize(256);
    dim3 gridSize2((num + blockSize.x - 1) / blockSize.x);
    kernelGround << <gridSize2, blockSize >> > (d_coeMatrix, num, d_numList);
    thrust::device_ptr<int> dev_ptr2(d_numList);
    highNum = thrust::reduce(dev_ptr2, dev_ptr2 + num);
    dim3 gridSize((num * num + blockSize.x - 1) / blockSize.x);
    switch (modId)
    {
    case 1:
        cutT = cutThreshold;
        return;
    case 2:
        cutT = cutThreshold * cutThreshold *cutThreshold;
        kernelFunction1 << <gridSize, blockSize >> > (d_coeMatrix, num);
        break;
    case 3:
        cutT = cutThreshold * (cutThreshold + 0.5);
        kernelFunction2 << <gridSize, blockSize >> > (d_coeMatrix, num);
        break;
    case 4:
        cutT = cutThreshold * (cutThreshold+1.0);
        kernelFunction3 << <gridSize, blockSize >> > (d_coeMatrix, num);
        break;
    }
}

void Operator::menu(const int funcId, const int iterTimes,const double iterDiff) {
    
    if (funcId == 1)
    {
        maxNum = iterTimes;
        minError = iterDiff;
    }
    if (funcId == 2)
    {
        maxNum = iterTimes;
        minError = 99999;
    }

    if (funcId == 3)
    {
        maxNum = 0;
        minError = iterDiff;
    }
    
}